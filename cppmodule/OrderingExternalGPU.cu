#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: jglaser

#include "OrderingExternalGPU.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif
#include <hip/hip_runtime.h>

/*! \file OrderingExternalGPU.cuh
    \brief Defines templated GPU kernel code for calculating the external forces.
*/

//! Kernel for calculating external forces
/*! This kernel is called to calculate the external forces on all N particles. Actual evaluation of the potentials and
    forces for each particle is handled via the template class \a evaluator.

    \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param d_pos device array of particle positions
    \param box Box dimensions used to implement periodic boundary conditions
    \param params per-type array of parameters for the potential

*/
__global__ void gpu_compute_ordering_external_forces_kernel(float4 *d_force,
                                               float *d_virial,
                                               const unsigned int virial_pitch,
                                               const unsigned int N,
                                               const Scalar4 *d_pos,
                                               const BoxDim box,
                                               const Scalar *order_parameters,
                                               const unsigned int n_wave, 
                                               const int3 *lattice_vectors,
                                               const Scalar interface_width,
                                               const unsigned int periodicity)
    {
    // start by identifying which particle we are to handle
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N)
        return;

    // read in the position of our particle.
    // (MEM TRANSFER: 16 bytes)
    Scalar4 posi = d_pos[idx];

    // initialize the force to 0
    Scalar3 force = make_scalar3(0.0, 0.0, 0.0);
    Scalar energy = Scalar(0.0);
    Scalar3 L = box.getL();

    Scalar3 Xi = make_scalar3((posi.x + (L.x/Scalar(2.0)))/(L.x), 
                              (posi.y + (L.y/Scalar(2.0)))/(L.y), 
                              (posi.z + (L.z/Scalar(2.0)))/(L.z));

    unsigned int typei = __float_as_int(posi.w);
    Scalar order_parameter = order_parameters[typei];

    Scalar cosine = Scalar(0.0);
    Scalar3 deriv = make_scalar3(0.0,0.0,0.0);
    for (unsigned int i = 0; i < n_wave; ++i) {
        Scalar3 q = make_scalar3(2.0*M_PI*lattice_vectors[i].x/L.x, 
                                 2.0*M_PI*lattice_vectors[i].y/L.y, 
                                 2.0*M_PI*lattice_vectors[i].z/L.z);
        Scalar3 qr = make_scalar3(2.0*M_PI*lattice_vectors[i].x,
                                  2.0*M_PI*lattice_vectors[i].y,
                                  2.0*M_PI*lattice_vectors[i].z);

        Scalar arg, q_length, clip_parameter, sine;
        arg = dot(Xi, qr);
        q_length = dot(q, L);
        if (lattice_vectors[i].x != 0 || lattice_vectors[i].y != 0 || lattice_vectors[i].z != 0) {
           clip_parameter = Scalar(1.0)/(interface_width*q_length);
        } else {
           clip_parameter = Scalar(0.0);
        }
        cosine += clip_parameter*cosf(arg);
        sine = -Scalar(1.0)*clip_parameter*sinf(arg);
        deriv = deriv + sine*q;
    }
    Scalar tanH = tanhf(cosine);
    
    energy = order_parameter*tanH;
    
    Scalar sechSq = (Scalar(1.0) - tanH*tanH);
    Scalar f = order_parameter*sechSq;
    force = f*deriv;

    // now that the force calculation is complete, write out the result)
    d_force[idx].x = force.x;
    d_force[idx].y = force.y;
    d_force[idx].z = force.z;
    d_force[idx].w = energy;

    for (unsigned int i = 0; i < 6; i++)
        d_virial[i] = Scalar(0.0);
    }

//! Kernel driver that computes lj forces on the GPU for LJForceComputeGPU
/*! \param external_potential_args Other arugments to pass onto the kernel
    \param d_params Parameters for the potential

    This is just a driver function for gpu_compute_external_forces(), see it for details.
*/
hipError_t gpu_compute_ordering_external_forces(float4 *d_force,
              float *d_virial,
              const unsigned int virial_pitch,
              const unsigned int N,
              const Scalar4 *d_pos,
              const BoxDim& box,
              const unsigned int block_size,
              const Scalar *d_order_parameters, 
              const unsigned int n_wave,
              const int3 *d_lattice_vectors,
              const Scalar interface_width,
              const unsigned int periodicity)
    {
    // setup the grid to run the kernel
    dim3 grid( N / block_size + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // bind the position texture
    gpu_compute_ordering_external_forces_kernel
           <<<grid, threads>>>(d_force, d_virial, virial_pitch, N, d_pos, box, d_order_parameters, n_wave, d_lattice_vectors, interface_width, periodicity);

    return hipSuccess;
    }

